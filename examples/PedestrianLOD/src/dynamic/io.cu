
/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <limits.h>
#include <algorithm>
#include <string>
#include <vector>



#ifdef _WIN32
#define strtok_r strtok_s
#endif

// include header
#include "header.h"

glm::vec3 agent_maximum;
glm::vec3 agent_minimum;

int fpgu_strtol(const char* str){
    return (int)strtol(str, NULL, 0);
}

unsigned int fpgu_strtoul(const char* str){
    return (unsigned int)strtoul(str, NULL, 0);
}

long long int fpgu_strtoll(const char* str){
    return strtoll(str, NULL, 0);
}

unsigned long long int fpgu_strtoull(const char* str){
    return strtoull(str, NULL, 0);
}

double fpgu_strtod(const char* str){
    return strtod(str, NULL);
}

float fgpu_atof(const char* str){
    return (float)atof(str);
}


//templated class function to read array inputs from supported types
template <class T>
void readArrayInput( T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: variable array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        array[i++] = (T)parseFunc(token);
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: variable array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

//templated class function to read array inputs from supported types
template <class T, class BASE_T, unsigned int D>
void readArrayInputVectorType( BASE_T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = "|";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent memory array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        //read vector type as an array
        T vec;
        readArrayInput<BASE_T>(parseFunc, token, (BASE_T*) &vec, D);
        array[i++] = vec;
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: Agent memory array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_agent_list* h_agents_default, xmachine_memory_agent_list* d_agents_default, int h_xmachine_memory_agent_default_count)
{
    PROFILE_SCOPED_RANGE("saveIterationData");
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_agents_default, d_agents_default, sizeof(xmachine_memory_agent_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying agent Agent default State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing iteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
    if(file == nullptr){
        printf("Error: Could not open file `%s` for output. Aborting.\n", data);
        exit(EXIT_FAILURE);
    }
    fputs("<states>\n<itno>", file);
    sprintf(data, "%i", iteration_number);
    fputs(data, file);
    fputs("</itno>\n", file);
    fputs("<environment>\n" , file);
    
    fputs("\t<TIME_SCALER>", file);
    sprintf(data, "%f", (*get_TIME_SCALER()));
    fputs(data, file);
    fputs("</TIME_SCALER>\n", file);
    fputs("\t<STEER_WEIGHT>", file);
    sprintf(data, "%f", (*get_STEER_WEIGHT()));
    fputs(data, file);
    fputs("</STEER_WEIGHT>\n", file);
    fputs("\t<AVOID_WEIGHT>", file);
    sprintf(data, "%f", (*get_AVOID_WEIGHT()));
    fputs(data, file);
    fputs("</AVOID_WEIGHT>\n", file);
    fputs("\t<COLLISION_WEIGHT>", file);
    sprintf(data, "%f", (*get_COLLISION_WEIGHT()));
    fputs(data, file);
    fputs("</COLLISION_WEIGHT>\n", file);
    fputs("\t<GOAL_WEIGHT>", file);
    sprintf(data, "%f", (*get_GOAL_WEIGHT()));
    fputs(data, file);
    fputs("</GOAL_WEIGHT>\n", file);
    fputs("\t<EYE_X>", file);
    sprintf(data, "%f", (*get_EYE_X()));
    fputs(data, file);
    fputs("</EYE_X>\n", file);
    fputs("\t<EYE_Y>", file);
    sprintf(data, "%f", (*get_EYE_Y()));
    fputs(data, file);
    fputs("</EYE_Y>\n", file);
    fputs("\t<EYE_Z>", file);
    sprintf(data, "%f", (*get_EYE_Z()));
    fputs(data, file);
    fputs("</EYE_Z>\n", file);
	fputs("</environment>\n" , file);

	//Write each agent agent to xml
	for (int i=0; i<h_xmachine_memory_agent_default_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>agent</name>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_agents_default->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_agents_default->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<velx>", file);
        sprintf(data, "%f", h_agents_default->velx[i]);
		fputs(data, file);
		fputs("</velx>\n", file);
        
		fputs("<vely>", file);
        sprintf(data, "%f", h_agents_default->vely[i]);
		fputs(data, file);
		fputs("</vely>\n", file);
        
		fputs("<steer_x>", file);
        sprintf(data, "%f", h_agents_default->steer_x[i]);
		fputs(data, file);
		fputs("</steer_x>\n", file);
        
		fputs("<steer_y>", file);
        sprintf(data, "%f", h_agents_default->steer_y[i]);
		fputs(data, file);
		fputs("</steer_y>\n", file);
        
		fputs("<height>", file);
        sprintf(data, "%f", h_agents_default->height[i]);
		fputs(data, file);
		fputs("</height>\n", file);
        
		fputs("<exit_no>", file);
        sprintf(data, "%d", h_agents_default->exit_no[i]);
		fputs(data, file);
		fputs("</exit_no>\n", file);
        
		fputs("<speed>", file);
        sprintf(data, "%f", h_agents_default->speed[i]);
		fputs(data, file);
		fputs("</speed>\n", file);
        
		fputs("<lod>", file);
        sprintf(data, "%d", h_agents_default->lod[i]);
		fputs(data, file);
		fputs("</lod>\n", file);
        
		fputs("<animate>", file);
        sprintf(data, "%f", h_agents_default->animate[i]);
		fputs(data, file);
		fputs("</animate>\n", file);
        
		fputs("<animate_dir>", file);
        sprintf(data, "%d", h_agents_default->animate_dir[i]);
		fputs(data, file);
		fputs("</animate_dir>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);

}

void readInitialStates(char* inputpath, xmachine_memory_agent_list* h_agents, int* h_xmachine_memory_agent_count)
{
    PROFILE_SCOPED_RANGE("readInitialStates");

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	const int bufferSize = 10000;
	char buffer[bufferSize];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_xagent, in_name, in_comment;
    int in_agent_x;
    int in_agent_y;
    int in_agent_velx;
    int in_agent_vely;
    int in_agent_steer_x;
    int in_agent_steer_y;
    int in_agent_height;
    int in_agent_exit_no;
    int in_agent_speed;
    int in_agent_lod;
    int in_agent_animate;
    int in_agent_animate_dir;
    
    /* tags for environment global variables */
    int in_env;
    int in_env_TIME_SCALER;
    
    int in_env_STEER_WEIGHT;
    
    int in_env_AVOID_WEIGHT;
    
    int in_env_COLLISION_WEIGHT;
    
    int in_env_GOAL_WEIGHT;
    
    int in_env_EYE_X;
    
    int in_env_EYE_Y;
    
    int in_env_EYE_Z;
    
	/* set agent count to zero */
	*h_xmachine_memory_agent_count = 0;
	
	/* Variables for initial state data */
	float agent_x;
	float agent_y;
	float agent_velx;
	float agent_vely;
	float agent_steer_x;
	float agent_steer_y;
	float agent_height;
	int agent_exit_no;
	float agent_speed;
	int agent_lod;
	float agent_animate;
	int agent_animate_dir;

    /* Variables for environment variables */
    float env_TIME_SCALER;
    float env_STEER_WEIGHT;
    float env_AVOID_WEIGHT;
    float env_COLLISION_WEIGHT;
    float env_GOAL_WEIGHT;
    float env_EYE_X;
    float env_EYE_Y;
    float env_EYE_Z;
    


	/* Initialise variables */
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
    in_comment = 0;
	in_tag = 0;
	in_itno = 0;
    in_env = 0;
    in_xagent = 0;
	in_name = 0;
	in_agent_x = 0;
	in_agent_y = 0;
	in_agent_velx = 0;
	in_agent_vely = 0;
	in_agent_steer_x = 0;
	in_agent_steer_y = 0;
	in_agent_height = 0;
	in_agent_exit_no = 0;
	in_agent_speed = 0;
	in_agent_lod = 0;
	in_agent_animate = 0;
	in_agent_animate_dir = 0;
    in_env_TIME_SCALER = 0;
    in_env_STEER_WEIGHT = 0;
    in_env_AVOID_WEIGHT = 0;
    in_env_COLLISION_WEIGHT = 0;
    in_env_GOAL_WEIGHT = 0;
    in_env_EYE_X = 0;
    in_env_EYE_Y = 0;
    in_env_EYE_Z = 0;
	//set all agent values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_agent_MAX; k++)
	{	
		h_agents->x[k] = 0;
		h_agents->y[k] = 0;
		h_agents->velx[k] = 0;
		h_agents->vely[k] = 0;
		h_agents->steer_x[k] = 0;
		h_agents->steer_y[k] = 0;
		h_agents->height[k] = 0;
		h_agents->exit_no[k] = 0;
		h_agents->speed[k] = 0;
		h_agents->lod[k] = 0;
		h_agents->animate[k] = 0;
		h_agents->animate_dir[k] = 0;
	}
	

	/* Default variables for memory */
    agent_x = 0;
    agent_y = 0;
    agent_velx = 0;
    agent_vely = 0;
    agent_steer_x = 0;
    agent_steer_y = 0;
    agent_height = 0;
    agent_exit_no = 0;
    agent_speed = 0;
    agent_lod = 0;
    agent_animate = 0;
    agent_animate_dir = 0;

    /* Default variables for environment variables */
    env_TIME_SCALER = 0;
    env_STEER_WEIGHT = 0;
    env_AVOID_WEIGHT = 0;
    env_COLLISION_WEIGHT = 0;
    env_GOAL_WEIGHT = 0;
    env_EYE_X = 0;
    env_EYE_Y = 0;
    env_EYE_Z = 0;
    
    
    // If no input path was specified, issue a message and return.
    if(inputpath[0] == '\0'){
        printf("No initial states file specified. Using default values.\n");
        return;
    }
    
    // Otherwise an input path was specified, and we have previously checked that it is (was) not a directory. 
    
	// Attempt to open the non directory path as read only.
	file = fopen(inputpath, "r");
    
    // If the file could not be opened, issue a message and return.
    if(file == nullptr)
    {
      printf("Could not open input file %s. Continuing with default values\n", inputpath);
      return;
    }
    // Otherwise we can iterate the file until the end of XML is reached.
    size_t bytesRead = 0;
    i = 0;
	while(reading==1)
	{
        // If I exceeds our buffer size we must abort
        if(i >= bufferSize){
            fprintf(stderr, "Error: XML Parsing failed Tag name or content too long (> %d characters)\n", bufferSize);
            exit(EXIT_FAILURE);
        }

		/* Get the next char from the file */
		c = (char)fgetc(file);

        // Check if we reached the end of the file.
        if(c == EOF){
            // Break out of the loop. This allows for empty files(which may or may not be)
            break;
        }
        // Increment byte counter.
        bytesRead++;

        /*If in a  comment, look for the end of a comment */
        if(in_comment){

            /* Look for an end tag following two (or more) hyphens.
               To support very long comments, we use the minimal amount of buffer we can. 
               If we see a hyphen, store it and increment i (but don't increment i)
               If we see a > check if we have a correct terminating comment
               If we see any other characters, reset i.
            */

            if(c == '-'){
                buffer[i] = c;
                i++;
            } else if(c == '>' && i >= 2){
                in_comment = 0;
                i = 0;
            } else {
                i = 0;
            }

            /*// If we see the end tag, check the preceding two characters for a close comment, if enough characters have been read for -->
            if(c == '>' && i >= 2 && buffer[i-1] == '-' && buffer[i-2] == '-'){
                in_comment = 0;
                buffer[0] = 0;
                i = 0;
            } else {
                // Otherwise just store it in the buffer so we can keep checking for close tags
                buffer[i] = c;
                i++;
            }*/
        }
		/* If the end of a tag */
		else if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;

			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
            if(strcmp(buffer, "environment") == 0) in_env = 1;
            if(strcmp(buffer, "/environment") == 0) in_env = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
            if(strcmp(buffer, "xagent") == 0) in_xagent = 1;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "agent") == 0)
				{
					if (*h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent agent exceeded whilst reading data\n", xmachine_memory_agent_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(EXIT_FAILURE);
					}
                    
					h_agents->x[*h_xmachine_memory_agent_count] = agent_x;//Check maximum x value
                    if(agent_maximum.x < agent_x)
                        agent_maximum.x = (float)agent_x;
                    //Check minimum x value
                    if(agent_minimum.x > agent_x)
                        agent_minimum.x = (float)agent_x;
                    
					h_agents->y[*h_xmachine_memory_agent_count] = agent_y;//Check maximum y value
                    if(agent_maximum.y < agent_y)
                        agent_maximum.y = (float)agent_y;
                    //Check minimum y value
                    if(agent_minimum.y > agent_y)
                        agent_minimum.y = (float)agent_y;
                    
					h_agents->velx[*h_xmachine_memory_agent_count] = agent_velx;
					h_agents->vely[*h_xmachine_memory_agent_count] = agent_vely;
					h_agents->steer_x[*h_xmachine_memory_agent_count] = agent_steer_x;
					h_agents->steer_y[*h_xmachine_memory_agent_count] = agent_steer_y;
					h_agents->height[*h_xmachine_memory_agent_count] = agent_height;
					h_agents->exit_no[*h_xmachine_memory_agent_count] = agent_exit_no;
					h_agents->speed[*h_xmachine_memory_agent_count] = agent_speed;
					h_agents->lod[*h_xmachine_memory_agent_count] = agent_lod;
					h_agents->animate[*h_xmachine_memory_agent_count] = agent_animate;
					h_agents->animate_dir[*h_xmachine_memory_agent_count] = agent_animate_dir;
					(*h_xmachine_memory_agent_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}



				/* Reset xagent variables */
                agent_x = 0;
                agent_y = 0;
                agent_velx = 0;
                agent_vely = 0;
                agent_steer_x = 0;
                agent_steer_y = 0;
                agent_height = 0;
                agent_exit_no = 0;
                agent_speed = 0;
                agent_lod = 0;
                agent_animate = 0;
                agent_animate_dir = 0;
                
                in_xagent = 0;
			}
			if(strcmp(buffer, "x") == 0) in_agent_x = 1;
			if(strcmp(buffer, "/x") == 0) in_agent_x = 0;
			if(strcmp(buffer, "y") == 0) in_agent_y = 1;
			if(strcmp(buffer, "/y") == 0) in_agent_y = 0;
			if(strcmp(buffer, "velx") == 0) in_agent_velx = 1;
			if(strcmp(buffer, "/velx") == 0) in_agent_velx = 0;
			if(strcmp(buffer, "vely") == 0) in_agent_vely = 1;
			if(strcmp(buffer, "/vely") == 0) in_agent_vely = 0;
			if(strcmp(buffer, "steer_x") == 0) in_agent_steer_x = 1;
			if(strcmp(buffer, "/steer_x") == 0) in_agent_steer_x = 0;
			if(strcmp(buffer, "steer_y") == 0) in_agent_steer_y = 1;
			if(strcmp(buffer, "/steer_y") == 0) in_agent_steer_y = 0;
			if(strcmp(buffer, "height") == 0) in_agent_height = 1;
			if(strcmp(buffer, "/height") == 0) in_agent_height = 0;
			if(strcmp(buffer, "exit_no") == 0) in_agent_exit_no = 1;
			if(strcmp(buffer, "/exit_no") == 0) in_agent_exit_no = 0;
			if(strcmp(buffer, "speed") == 0) in_agent_speed = 1;
			if(strcmp(buffer, "/speed") == 0) in_agent_speed = 0;
			if(strcmp(buffer, "lod") == 0) in_agent_lod = 1;
			if(strcmp(buffer, "/lod") == 0) in_agent_lod = 0;
			if(strcmp(buffer, "animate") == 0) in_agent_animate = 1;
			if(strcmp(buffer, "/animate") == 0) in_agent_animate = 0;
			if(strcmp(buffer, "animate_dir") == 0) in_agent_animate_dir = 1;
			if(strcmp(buffer, "/animate_dir") == 0) in_agent_animate_dir = 0;
			
            /* environment variables */
            if(strcmp(buffer, "TIME_SCALER") == 0) in_env_TIME_SCALER = 1;
            if(strcmp(buffer, "/TIME_SCALER") == 0) in_env_TIME_SCALER = 0;
			if(strcmp(buffer, "STEER_WEIGHT") == 0) in_env_STEER_WEIGHT = 1;
            if(strcmp(buffer, "/STEER_WEIGHT") == 0) in_env_STEER_WEIGHT = 0;
			if(strcmp(buffer, "AVOID_WEIGHT") == 0) in_env_AVOID_WEIGHT = 1;
            if(strcmp(buffer, "/AVOID_WEIGHT") == 0) in_env_AVOID_WEIGHT = 0;
			if(strcmp(buffer, "COLLISION_WEIGHT") == 0) in_env_COLLISION_WEIGHT = 1;
            if(strcmp(buffer, "/COLLISION_WEIGHT") == 0) in_env_COLLISION_WEIGHT = 0;
			if(strcmp(buffer, "GOAL_WEIGHT") == 0) in_env_GOAL_WEIGHT = 1;
            if(strcmp(buffer, "/GOAL_WEIGHT") == 0) in_env_GOAL_WEIGHT = 0;
			if(strcmp(buffer, "EYE_X") == 0) in_env_EYE_X = 1;
            if(strcmp(buffer, "/EYE_X") == 0) in_env_EYE_X = 0;
			if(strcmp(buffer, "EYE_Y") == 0) in_env_EYE_Y = 1;
            if(strcmp(buffer, "/EYE_Y") == 0) in_env_EYE_Y = 0;
			if(strcmp(buffer, "EYE_Z") == 0) in_env_EYE_Z = 1;
            if(strcmp(buffer, "/EYE_Z") == 0) in_env_EYE_Z = 0;
			

			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;

			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else if (in_xagent)
			{
				if(in_agent_x){
                    agent_x = (float) fgpu_atof(buffer); 
                }
				if(in_agent_y){
                    agent_y = (float) fgpu_atof(buffer); 
                }
				if(in_agent_velx){
                    agent_velx = (float) fgpu_atof(buffer); 
                }
				if(in_agent_vely){
                    agent_vely = (float) fgpu_atof(buffer); 
                }
				if(in_agent_steer_x){
                    agent_steer_x = (float) fgpu_atof(buffer); 
                }
				if(in_agent_steer_y){
                    agent_steer_y = (float) fgpu_atof(buffer); 
                }
				if(in_agent_height){
                    agent_height = (float) fgpu_atof(buffer); 
                }
				if(in_agent_exit_no){
                    agent_exit_no = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_speed){
                    agent_speed = (float) fgpu_atof(buffer); 
                }
				if(in_agent_lod){
                    agent_lod = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_animate){
                    agent_animate = (float) fgpu_atof(buffer); 
                }
				if(in_agent_animate_dir){
                    agent_animate_dir = (int) fpgu_strtol(buffer); 
                }
				
            }
            else if (in_env){
            if(in_env_TIME_SCALER){
              
                    env_TIME_SCALER = (float) fgpu_atof(buffer);
                    
                    set_TIME_SCALER(&env_TIME_SCALER);
                  
              }
            if(in_env_STEER_WEIGHT){
              
                    env_STEER_WEIGHT = (float) fgpu_atof(buffer);
                    
                    set_STEER_WEIGHT(&env_STEER_WEIGHT);
                  
              }
            if(in_env_AVOID_WEIGHT){
              
                    env_AVOID_WEIGHT = (float) fgpu_atof(buffer);
                    
                    set_AVOID_WEIGHT(&env_AVOID_WEIGHT);
                  
              }
            if(in_env_COLLISION_WEIGHT){
              
                    env_COLLISION_WEIGHT = (float) fgpu_atof(buffer);
                    
                    set_COLLISION_WEIGHT(&env_COLLISION_WEIGHT);
                  
              }
            if(in_env_GOAL_WEIGHT){
              
                    env_GOAL_WEIGHT = (float) fgpu_atof(buffer);
                    
                    set_GOAL_WEIGHT(&env_GOAL_WEIGHT);
                  
              }
            if(in_env_EYE_X){
              
                    env_EYE_X = (float) fgpu_atof(buffer);
                    
                    set_EYE_X(&env_EYE_X);
                  
              }
            if(in_env_EYE_Y){
              
                    env_EYE_Y = (float) fgpu_atof(buffer);
                    
                    set_EYE_Y(&env_EYE_Y);
                  
              }
            if(in_env_EYE_Z){
              
                    env_EYE_Z = (float) fgpu_atof(buffer);
                    
                    set_EYE_Z(&env_EYE_Z);
                  
              }
            
            }
		/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
            // Check if we are a comment, when we are in a tag and buffer[0:2] == "!--"
            if(i == 2 && c == '-' && buffer[1] == '-' && buffer[0] == '!'){
                in_comment = 1;
                // Reset the buffer and i.
                buffer[0] = 0;
                i = 0;
            }

            // Store the character and increment the counter
            buffer[i] = c;
            i++;

		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
    // If no bytes were read, raise a warning.
    if(bytesRead == 0){
        fprintf(stdout, "Warning: %s is an empty file\n", inputpath);
        fflush(stdout);
    }

    // If the in_comment flag is still marked, issue a warning.
    if(in_comment){
        fprintf(stdout, "Warning: Un-terminated comment in %s\n", inputpath);
        fflush(stdout);
    }    

	/* Close the file */
	fclose(file);
}

glm::vec3 getMaximumBounds(){
    return agent_maximum;
}

glm::vec3 getMinimumBounds(){
    return agent_minimum;
}


/* Methods to load static networks from disk */
